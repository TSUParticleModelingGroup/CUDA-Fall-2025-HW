// nvcc GlobalVsStaticVsLocal.cu -o temp

// temp1 is a normal local variable. It gets reset every time the function is called.

// temp2 is a static local variable. It gets set the first time the function is call but is remembered and not
// reset in subsaquent calls.

// temp3 is a global variable. The static variable acts like a global but can "NOT" be seen from other functions
// so it is safer in that other functions cannot adjust it by mistake. But it can be dangerous if you mistakenly
// think it is going to always be set at each function call. 

// temp4 is an external variable meaning it is a global variable but is defined in another file.


#include <hip/hip_runtime.h>
#include <stdio.h>
// Prototypes 
void incrementNumber();

// Globals
int temp3 = 0;

void incrementNumber()
{
	int temp1 = 0;
	static int temp2 = 0;
	
	temp1++;
	temp2++;
	temp3++;
	printf(" temp1 = %d \n", temp1);
	printf(" temp2 = %d \n", temp2);
	printf(" temp3 = %d \n", temp3);
	printf(" *******************************************\n");
}

int main()
{
	incrementNumber();
	incrementNumber();
	incrementNumber();
	incrementNumber();
	return 0;
}


// https://www.youtube.com/watch?v=ZLhnZAbcs2s
