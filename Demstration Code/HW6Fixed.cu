#include "hip/hip_runtime.h"
// Name: Fixed
// Simple Julia CPU.
// nvcc HW6Fixed.cu -o temp -lglut -lGL
// glut and GL are openGL libraries.
/*
 What to do:
 This code displays a simple Julia fractal using the CPU.
 Rewrite the code so that it uses the GPU to create the fractal. 
 Keep the window at 1024 by 1024.
*/

/*
 What was done:
 Took the color pixels function to the GPU.
*/

// Include files
#include <stdio.h>
#include <GL/glut.h>

// Defines
#define MAXMAG 10.0 // If you grow larger than this, we assume that you have escaped.
#define MAXITERATIONS 200 // If you have not escaped after this many attempts, we assume you are not going to escape.
#define A  -0.824	//Real part of C
#define B  -0.1711	//Imaginary part of C

// Global variables
unsigned int WindowWidth = 1024;
unsigned int WindowHeight = 1024;

float XMin = -2.0;
float XMax =  2.0;
float YMin = -2.0;
float YMax =  2.0;

// Function prototypes
void cudaErrorCheck(const char*, int);
__global__ void colorPixels(float, float, float, float, float);

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

__global__ void colorPixels(float *pixels, float xMin, float yMin, float dx, float dy) 
{
	float x,y,mag,tempX;
	int count, id;
	
	int maxCount = MAXITERATIONS;
	float maxMag = MAXMAG;
	
	
	//Getting the offset into the pixel buffer. 
	//We need the 3 because each pixel has a red, green, and blue value.
	id = 3*(threadIdx.x + blockDim.x*blockIdx.x);
	
	//Asigning each thread its x and y value of its pixel.
	x = xMin + dx*threadIdx.x;
	y = yMin + dy*blockIdx.x;
	
	count = 0;
	mag = sqrt(x*x + y*y);;
	while (mag < maxMag && count < maxCount) 
	{
		//We will be changing the x but we need its old value to find y.	
		tempX = x; 
		x = x*x - y*y + A;
		y = (2.0 * tempX * y) + B;
		mag = sqrt(x*x + y*y);
		count++;
	}
	
	//Setting the red value
	if(count < maxCount) //It excaped
	{
		pixels[id] = 0.0;
	}
	else //It Stuck around
	{
		pixels[id] = 1.0;
	}
	//Setting the green
	pixels[id+1] = 0.0;
	//Setting the blue 
	pixels[id+2] = 0.0;
}

void display(void) 
{ 
	dim3 blockSize, gridSize;
	float *pixelsCPU, *pixelsGPU; 
	float stepSizeX, stepSizeY;
	
	//We need the 3 because each pixel has a red, green, and blue value.
	pixelsCPU = (float *)malloc(WindowWidth*WindowHeight*3*sizeof(float));
	hipMalloc(&pixelsGPU,WindowWidth*WindowHeight*3*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	
	stepSizeX = (XMax - XMin)/((float)WindowWidth);
	stepSizeY = (YMax - YMin)/((float)WindowHeight);
	
	//Threads in a block
	if(WindowWidth > 1024)
	{
	 	printf("The window width is too large to run with this program\n");
	 	printf("The window width width must be less than 1024.\n");
	 	printf("Good Bye and have a nice day!\n");
	 	exit(0);
	}
	blockSize.x = 1024; //WindowWidth;
	blockSize.y = 1;
	blockSize.z = 1;
	
	//Blocks in a grid
	gridSize.x = WindowHeight;
	gridSize.y = 1;
	gridSize.z = 1;
	
	colorPixels<<<gridSize, blockSize>>>(pixelsGPU, XMin, YMin, stepSizeX, stepSizeY);
	cudaErrorCheck(__FILE__, __LINE__);
	
	//Copying the pixels that we just colored back to the CPU.
	hipMemcpyAsync(pixelsCPU, pixelsGPU, WindowWidth*WindowHeight*3*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	//Putting pixels on the screen.
	glDrawPixels(WindowWidth, WindowHeight, GL_RGB, GL_FLOAT, pixelsCPU); 
	glFlush(); 
}

int main(int argc, char** argv)
{ 
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(WindowWidth, WindowHeight);
	glutCreateWindow("Fractals--Man--Fractals");
   	glutDisplayFunc(display);
   	glutMainLoop();
}



