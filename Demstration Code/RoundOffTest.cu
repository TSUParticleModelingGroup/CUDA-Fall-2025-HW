//Playing around with roundoff
//nvcc RoundOffTest.cu -o temp

#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
	float a;
	float b = 0.1;
	float c;
	double d;
	
	printf("\n");
	
	a = 10000000.0;
	printf("a = %f, b = %f a+b = %f\n", a, b, a+b);
	
	a = 1000000.0;
	printf("a = %f, b = %f a+b = %f\n", a, b, a+b);
	
	a = 100000.0;
	printf("a = %f, b = %f a+b = %f\n", a, b, a+b);
	
	a = 10000.0;
	printf("a = %f, b = %f a+b = %f\n", a, b, a+b);
	
	a = 1000.0;
	printf("a = %f, b = %f a+b = %f\n", a, b, a+b);
	
	a = 100.0;
	printf("a = %f, b = %f a+b = %f\n", a, b, a+b);
	
	a = 10.0;
	printf("a = %f, b = %f a+b = %f\n", a, b, a+b);
	
	a = 1.0;
	printf("a = %f, b = %f a+b = %f\n", a, b, a+b);
	
	printf("\n");
	a = 10000000.0;
	c = a;
	for(int i = 0; i < 1000000; i++)
	{
		c = c + b;
	}
	printf("float:  a = %f, b = %f a+1000000*b = %f\n", a, b, c);
	
	d = a;
	for(int i = 0; i < 1000000; i++)
	{
		d = d + (double)b;
	}
	printf("double: a = %f, b = %f a+1000000*b = %f\n", a, b, d);
	
	printf("\n");
	a = 1000000.0;
	c = a;
	for(int i = 0; i < 1000000; i++)
	{
		c = c + b;
	}
	printf("float:  a = %f, b = %f a+1000000*b = %f\n", a, b, c);
	
	d = a;
	for(int i = 0; i < 1000000; i++)
	{
		d = d + (double)b;
	}
	printf("double: a = %f, b = %f a+1000000*b = %f\n", a, b, d);
	
	printf("\n");
	a = 100000.0;
	c = a;
	for(int i = 0; i < 1000000; i++)
	{
		c = c + b;
	}
	printf("float:  a = %f, b = %f a+1000000*b = %f\n", a, b, c);
	
	d = a;
	for(int i = 0; i < 1000000; i++)
	{
		d = d + (double)b;
	}
	printf("double: a = %f, b = %f a+1000000*b = %f\n", a, b, d);
	
	printf("\n");
	a = 10000.0;
	c = a;
	for(int i = 0; i < 1000000; i++)
	{
		c = c + b;
	}
	printf("float:  a = %f, b = %f a+1000000*b = %f\n", a, b, c);
	
	d = a;
	for(int i = 0; i < 1000000; i++)
	{
		d = d + (double)b;
	}
	printf("double: a = %f, b = %f a+1000000*b = %f\n", a, b, d);
	
	printf("\n");
	a = 1000.0;
	c = a;
	for(int i = 0; i < 1000000; i++)
	{
		c = c + b;
	}
	printf("float:  a = %f, b = %f a+1000000*b = %f\n", a, b, c);
	
	d = a;
	for(int i = 0; i < 1000000; i++)
	{
		d = d + (double)b;
	}
	printf("double: a = %f, b = %f a+1000000*b = %f\n", a, b, d);
	
	printf("\n");
	a = 100.0;
	c = a;
	for(int i = 0; i < 1000000; i++)
	{
		c = c + b;
	}
	printf("float:  a = %f, b = %f a+1000000*b = %f\n", a, b, c);
	
	d = a;
	for(int i = 0; i < 1000000; i++)
	{
		d = d + (double)b;
	}
	printf("double: a = %f, b = %f a+1000000*b = %f\n", a, b, d);
	
	printf("\n");
	a = 10.0;
	c = a;
	for(int i = 0; i < 1000000; i++)
	{
		c = c + b;
	}
	printf("float:  a = %f, b = %f a+1000000*b = %f\n", a, b, c);
	
	d = a;
	for(int i = 0; i < 1000000; i++)
	{
		d = d + (double)b;
	}
	printf("double: a = %f, b = %f a+1000000*b = %f\n", a, b, d);
	
	printf("\n");
	a = 1.0;
	c = a;
	for(int i = 0; i < 1000000; i++)
	{
		c = c + b;
	}
	printf("float:  a = %f, b = %f a+1000000*b = %f\n", a, b, c);
	
	d = a;
	for(int i = 0; i < 1000000; i++)
	{
		d = d + (double)b;
	}
	printf("double: a = %f, b = %f a+1000000*b = %f\n", a, b, d);
	
	printf("\n");
	c = 0.0;
	for(int i = 0; i < 100000000; i++)
	{
		c = c + b;
	}
	for(int i = 0; i < 100000000; i++)
	{
		c = c - b;
	}
	printf("float+-:  c = %f\n",c);
	
	printf("\n");
	c = 0.0;
	for(int i = 0; i < 100000000; i++)
	{
		c = c - b;
	}
	for(int i = 0; i < 100000000; i++)
	{
		c = c + b;
	}
	printf("float-+:  c = %f\n",c);
	
	
	return(0);
}
