// nvcc -c otherFile.cu -o otherFile.o
// nvcc -c VariableDemo.cu -o VariableDemo.o -lglut 
// nvcc otherFile.o VariableDemo.o -o tempProgram -lglut

// temp1 is a normal local variable. It gets reset every time the function is called.

// temp2 is a static local variable. It gets set the first time the function is call but is remembered and not
// reset in subsaquent calls.

// temp3 is a global variable. The static variable acts like a global but can not be seen from other functions
// so it is safer in that other functions cannot adjust it by mistake. But it can be dangerous if you mistakenly
// think it is going to always be set at each function call. 

// temp4 is an external variable meaning it is a global variable but is defined in another file.


#include <hip/hip_runtime.h>
#include <stdio.h>

void externExampleFunction();

// Globals
int temp3 = 0;
int temp4 = 5;

void incrementNumber()
{
	int temp1 = 0;
	static int temp2 = 0; // Acts like a global but is local. Can be safer because no other function can change it.
	
	temp1++;
	temp2++;
	temp3++;
	printf(" temp1 = %d \n", temp1);
	printf(" temp2 = %d \n", temp2);
	printf(" temp3 = %d \n", temp3);
	
	externExampleFunction();
}

int main()
{
	incrementNumber();
	incrementNumber();
	incrementNumber();
	incrementNumber();
	return 0;
}


// https://www.youtube.com/watch?v=ZLhnZAbcs2s
